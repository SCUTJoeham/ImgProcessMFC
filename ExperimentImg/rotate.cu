#include "hip/hip_runtime.h"
#ifndef  __ROTATE_CU_
#define  __ROTATE_CU_

#include "hip/hip_runtime.h"
#include ""
#include <opencv2\opencv.hpp>
#include <time.h>
#include <iostream>

inline void checkCudaErrors(hipError_t err) //cuda error handle function
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA Runtime API error:%s.\n", hipGetErrorString(err));
		return;
	}
}

__global__ void Rotate(uchar* _src_dev, uchar * _dst_dev, int _src_step, int _dst_step,
	int _src_rows, int _src_cols, int _dst_rows, int _dst_cols, double rr)
{
	int i = blockIdx.x;
	int j = blockIdx.y;

	double x = (int)((i - _dst_rows / 2)*cos(rr) - (j - _dst_cols / 2)*sin(rr) + 0.5);
	double y = (int)((i - _dst_rows / 2)*sin(rr) + (j - _dst_cols / 2)*cos(rr) + 0.5);

	x += _src_rows / 2;
	y += _src_cols / 2;

	int px = (int)x;
	int py = (int)y;
	double w_x[4], w_y[4];

	const double cubic_a = -0.5;
	double stemp_x[4], stemp_y[4];
	stemp_x[0] = 1 + (x - px); stemp_x[1] = x - px; stemp_x[2] = 1 - (x - px); stemp_x[3] = 2 - (x - px);
	w_x[0] = cubic_a * abs(stemp_x[0] * stemp_x[0] * stemp_x[0]) - 5 * cubic_a*stemp_x[0] * stemp_x[0] + 8 * cubic_a*abs(stemp_x[0]) - 4 * cubic_a;
	w_x[1] = (cubic_a + 2)*abs(stemp_x[1] * stemp_x[1] * stemp_x[1]) - (cubic_a + 3)*stemp_x[1] * stemp_x[1] + 1;
	w_x[2] = (cubic_a + 2)*abs(stemp_x[2] * stemp_x[2] * stemp_x[2]) - (cubic_a + 3)*stemp_x[2] * stemp_x[2] + 1;
	w_x[3] = cubic_a * abs(stemp_x[3] * stemp_x[3] * stemp_x[3]) - 5 * cubic_a*stemp_x[3] * stemp_x[3] + 8 * cubic_a*abs(stemp_x[3]) - 4 * cubic_a;

	stemp_y[0] = 1 + (y - py); stemp_y[1] = y - py; stemp_y[2] = 1 - (y - py); stemp_y[3] = 2 - (y - py);
	w_y[0] = cubic_a * abs(stemp_y[0] * stemp_y[0] * stemp_y[0]) - 5 * cubic_a*stemp_y[0] * stemp_y[0] + 8 * cubic_a*abs(stemp_y[0]) - 4 * cubic_a;
	w_y[1] = (cubic_a + 2)*abs(stemp_y[1] * stemp_y[1] * stemp_y[1]) - (cubic_a + 3)*stemp_y[1] * stemp_y[1] + 1;
	w_y[2] = (cubic_a + 2)*abs(stemp_y[2] * stemp_y[2] * stemp_y[2]) - (cubic_a + 3)*stemp_y[2] * stemp_y[2] + 1;
	w_y[3] = cubic_a * abs(stemp_y[3] * stemp_y[3] * stemp_y[3]) - 5 * cubic_a*stemp_y[3] * stemp_y[3] + 8 * cubic_a*abs(stemp_y[3]) - 4 * cubic_a;

	double tmp_r = 0.0, tmp_g = 0.0, tmp_b = 0.0;
	for (int s = 0; s <= 3; s++) {
		for (int t = 0; t <= 3; t++) {
			int temp_x = px + s - 1;
			int temp_y = py + t - 1;
			if (temp_x > -1 && temp_y > -1 && temp_x < _src_rows && temp_y < _src_cols) {
				tmp_r += *(_src_dev + temp_x * _src_step + 3 * temp_y + 2) * w_x[s] * w_y[t];
				tmp_g += *(_src_dev + temp_x * _src_step + 3 * temp_y + 1) * w_x[s] * w_y[t];
				tmp_b += *(_src_dev + temp_x * _src_step + 3 * temp_y) * w_x[s] * w_y[t];
			}
		}
	}
	if (x >= _src_rows || y >= _src_cols || x <= 0 || y <= 0)
	{
		*(_dst_dev + i * _dst_step + 3 * j + 2) = 0;
		*(_dst_dev + i * _dst_step + 3 * j + 1) = 0;
		*(_dst_dev + i * _dst_step + 3 * j) = 0;
	}
	else {
		*(_dst_dev + i * _dst_step + 3 * j + 2) = (uchar)tmp_r;
		*(_dst_dev + i * _dst_step + 3 * j + 1) = (uchar)tmp_g;
		*(_dst_dev + i * _dst_step + 3 * j) = (uchar)tmp_b;
	}
	
}

extern "C" void Rotate_host(const cv::Mat &src, cv::Mat &dst, const cv::Size &dst_s, double r_angel)
{
	uchar *src_data = src.data;
	int srcWidth = src.cols;
	int srcHeight = src.rows;
	int src_step = src.step;
	int dst_step = dst.step;
	uchar *src_dev, *dst_dev;

	const double PI = acos(-1);
	double rotate_radian = r_angel * PI / 180;

	checkCudaErrors(hipMalloc((void**)&src_dev, 3 * srcWidth * srcHeight * sizeof(uchar)));
	checkCudaErrors(hipMalloc((void**)&dst_dev, 3 * dst_s.width * dst_s.height * sizeof(uchar)));

	checkCudaErrors(hipMemcpy(src_dev, src_data, 3 * srcWidth * srcHeight * sizeof(uchar), hipMemcpyHostToDevice));

	dim3 grid(dst_s.height, dst_s.width);

	Rotate << < grid, 1 >> > (src_dev, dst_dev, src_step, dst_step, srcHeight, srcWidth, dst_s.height, dst_s.width, rotate_radian);

	checkCudaErrors(hipMemcpy(dst.data, dst_dev, 3 * dst_s.width * dst_s.height * sizeof(uchar), hipMemcpyDeviceToHost));

	hipFree(src_dev);
	hipFree(dst_dev);

}

#endif // ! __RESIZE_KERNEL_CU_