#include "hip/hip_runtime.h"
#ifndef  __FOURIERTRANS_CU_
#define  __FOURIERTRANS_CU_

#include "hip/hip_runtime.h"
#include ""
#include <opencv2\opencv.hpp>
#include <time.h>
#include <iostream>

inline void checkCudaErrors(hipError_t err) //cuda error handle function
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA Runtime API error:%s.\n", hipGetErrorString(err));
		return;
	}
}

__global__ void FourierTrans(uchar* _src_dev, uchar * _dst_dev, int _src_step, int _dst_step,
	int _src_rows, int _src_cols, int _dst_rows, int _dst_cols)
{
	int i = blockIdx.x;
	int j = blockIdx.y;

	const double PI = 3.1415926535;
	const int fourier_factor = 14;
	double real = 0.0, imag = 0.0;

	for (int x = 0; x < _src_rows; x++)
	{
		for (int y = 0; y < _src_cols; y++)
		{
			uchar pixel_r = *(_src_dev + x * _src_step + 3 * y + 2);
			uchar pixel_g = *(_src_dev + x * _src_step + 3 * y + 1);
			uchar pixel_b = *(_src_dev + x * _src_step + 3 * y);
			double gray = 0.299 * pixel_r + 0.587 * pixel_g + 0.114 * pixel_b;
			if ((x + y) & 1) // centralize
				gray = -gray;
			double A = 2 * PI * ((double)i * (double)x / (double)_src_rows + (double)j * (double)y / (double)_src_cols);
			real += gray * cos(A);
			imag -= gray * sin(A);
		}
	}
	double mag = sqrt(real * real + imag * imag);
	mag = fourier_factor * log(mag + 1);
	mag < 0 ? mag = 0 : (mag > 255 ? mag = 255 : mag = mag);

	*(_dst_dev + i * _dst_step + 3 * j + 2) = (uchar)mag;
	*(_dst_dev + i * _dst_step + 3 * j + 1) = (uchar)mag;
	*(_dst_dev + i * _dst_step + 3 * j) = (uchar)mag;

}

extern "C" void FourierTrans_host(const cv::Mat &src, cv::Mat &dst, const cv::Size &dst_s)
{
	uchar *src_data = src.data;
	int srcWidth = src.cols;
	int srcHeight = src.rows;
	int src_step = src.step;
	int dst_step = dst.step;
	uchar *src_dev, *dst_dev;


	checkCudaErrors(hipMalloc((void**)&src_dev, 3 * srcWidth * srcHeight * sizeof(uchar)));
	checkCudaErrors(hipMalloc((void**)&dst_dev, 3 * dst_s.width * dst_s.height * sizeof(uchar)));

	checkCudaErrors(hipMemcpy(src_dev, src_data, 3 * srcWidth * srcHeight * sizeof(uchar), hipMemcpyHostToDevice));

	dim3 grid(dst_s.height, dst_s.width);

	FourierTrans << < grid, 1 >> > (src_dev, dst_dev, src_step, dst_step, srcHeight, srcWidth, dst_s.height, dst_s.width);

	checkCudaErrors(hipMemcpy(dst.data, dst_dev, 3 * dst_s.width * dst_s.height * sizeof(uchar), hipMemcpyDeviceToHost));

	hipFree(src_dev);
	hipFree(dst_dev);

}

#endif // ! __RESIZE_KERNEL_CU_